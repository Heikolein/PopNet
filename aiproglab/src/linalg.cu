#include <hipblas.h>
#include <hip/hip_runtime.h>

void check_ptr_on_device(void** ptrs, int num) {
    for (int i = 0; i < num; i++) {
        hipPointerAttribute_t attributes;
        hipError_t error = hipPointerGetAttributes(&attributes, ptrs[i]);
        
        if (error == hipSuccess) {
            if (attributes.memoryType == hipMemoryTypeHost) {
                throw std::runtime_error("Data not on GPU");
            }
        } else {
            std::cerr << "Failed to get pointer attributes for pointer " << i << ": " << hipGetErrorString(error) << std::endl;
            throw std::runtime_error("");
        }
    }
}

// X(m*n) = A(m*k) * B(k*n) + C(m*n)
__device__ void gemm_gpu(const float *A, const float *B, float *C, float *X, const float beta, const int m, const int k, const int n) {
    int lda = m, ldb = k, ldc = m;
    const float alf = 1;
    const float *alpha = &alf;
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS initialization failed" << std::endl;
        return EXIT_FAILURE;
    }
    // Do the actual multiplication
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, alpha, B, n, A, k, beta, C, n);
    hipDeviceSynchronize();
    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, ldc, 1, C, 0, C, X, n);
    hipDeviceSynchronize();
    // Destroy the handle
    hipblasDestroy(handle);
}
