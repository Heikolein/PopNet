#include "hip/hip_runtime.h"
#include <hiprand.h>

// Fill the matrix with random numbers on GPU
void matrix_init(float *A, int rows, int cols) {
    // Create a pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    // Set the seed for the random number generator using the system clock
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());
    // Fill the array with random numbers on the device
    hiprandGenerateUniform(prng, A, rows * cols);
    hiprandDestroyGenerator(prng);
}
